#include "hip/hip_runtime.h"
#include <opencv2/core.hpp>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/cudev/common.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>

#include <iostream>

// CUDAカーネル
__global__ void inversionGpu
(
	const cv::cuda::PtrStepSz<uchar> src,
	cv::cuda::PtrStepSz<uchar> dst
)
{
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	if((y >= 0) && (y < src.rows))
	{
		if((x >= 0) && (x < src.cols))
		{
			dst.ptr(y)[x] = (255 - src.ptr(y)[x]);
		}
	}
}

void launchInversionGpu
(
	cv::cuda::GpuMat& src,
	cv::cuda::GpuMat& dst
)
{
	const dim3 block(32, 32);
	const dim3 grid(cv::cudev::divUp(dst.cols, block.x), cv::cudev::divUp(dst.rows, block.y));

	// CUDAカーネル起動
	inversionGpu<<<grid, block>>>(src, dst);

	CV_CUDEV_SAFE_CALL(hipGetLastError());
	CV_CUDEV_SAFE_CALL(hipDeviceSynchronize());
}

int main(int argc, char *argv[])
{
	cv::Mat src = cv::imread("lena.jpg", cv::IMREAD_GRAYSCALE);
	if (src.empty())
	{
		std::cerr << "could not load image." << std::endl;
		return -1;
	}

	cv::cuda::GpuMat d_src(src);
	cv::cuda::GpuMat d_dst(d_src.size(), d_src.type());
	launchInversionGpu(d_src, d_dst);

	cv::Mat dst;
	d_dst.download(dst);

	cv::imshow("dst", dst);
	cv::waitKey(0);
	cv::destroyAllWindows();

	return 0;
}
